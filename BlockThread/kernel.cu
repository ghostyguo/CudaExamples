#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void hello()
{
	printf("This is thread %d in block %d\n", threadIdx.x, blockIdx.x);
}

int main()
{
	int blockSize =4, threadSize = 3;

    // running in parallel
	hello << <blockSize, threadSize >> > ();

	//force the printf to flush
	hipDeviceSynchronize();
	getchar(); //wait keypressed

    return 0;
}
