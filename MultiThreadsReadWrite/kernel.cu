#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <stdio.h>

#define ThreadSize 1000
#define BlockSize  10000
#define ArraySize 10


//int atomicAdd(int* address, int val);
__global__ void incKernel(int *a)
{
	int i = (blockIdx.x*blockDim.x + threadIdx.x) % ArraySize;
	a[i] = a[i] + 1;
	//atomicAdd(&a[i], 1);
}

int main()
{
	int host_a[ArraySize];
	int *dev_a = 0;
	float elapsedTime;

	// setup performance meter from CUDA ----------
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipSetDevice(0);
	hipMalloc((void**)&dev_a, ArraySize * sizeof(int));

	for (int run = 0; run < 10; run++) {

		hipMemset(dev_a, 0, ArraySize * sizeof(int));		//clear 

		hipEventRecord(start, 0); //keep start time
		incKernel << <BlockSize, ThreadSize >> > (dev_a);	//calculate
		hipEventRecord(stop, 0); //keep stop time
		hipEventSynchronize(stop); //wait stop event		
		hipEventElapsedTime(&elapsedTime, start, stop);	

		hipMemcpy(host_a, dev_a, ArraySize * sizeof(int), hipMemcpyDeviceToHost);
		//Print result
		printf("run {%d}: ",run);
		for (int i = 0; i < ArraySize; i++) {
			printf("%d ", host_a[i]);
		}
		printf(" t=%f\n",elapsedTime);
	}
	//hipDeviceSynchronize();
	getchar();

	hipFree(dev_a);
	return 0;

}
