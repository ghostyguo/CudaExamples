#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#define BlockSize	8
#define ThreadSize	1024
#define ArraySize	(BlockSize*ThreadSize)

__device__ void __syncthreads();
__global__ void globalReduceBlockSum(int *b, int *a)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	for (int d = blockDim.x / 2; d > 0; d >>= 1)
	{
		if (threadIdx.x < d)
		{
			a[id] += a[id + d];
		}		
		__syncthreads();
	}

	if (threadIdx.x == 0) {
		b[blockIdx.x] = a[id]; 
	}
}

__global__ void sharedReduceBlockSum(int *b, int *a)
{
	__shared__ int x[ThreadSize];

	int id = blockIdx.x * blockDim.x + threadIdx.x;
	x[threadIdx.x] = a[id]; //copy to shared memory of block
	__syncthreads(); //wait all threads copy complete

	for (int d = blockDim.x / 2; d > 0; d >>= 1)
	{
		if (threadIdx.x < d)
		{
			x[threadIdx.x] += x[threadIdx.x + d];
		}
		__syncthreads();
	}

	if (threadIdx.x == 0) {
		b[blockIdx.x] = x[0];
	}
}

int main()
{
	int host_a[ArraySize];
	int host_b[BlockSize];
	int *dev_a = 0;
	int *dev_b = 0;
	int sum = 0;
	float elapsedTime;

	// setup performance meter from CUDA ----------
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipSetDevice(0);

	hipMalloc((void**)&dev_a, ArraySize * sizeof(int));
	for (int i = 0; i <  ArraySize; i++)
		host_a[i] = i+1;
	hipMemcpy(dev_a, host_a, ArraySize * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc((void**)&dev_b, BlockSize * sizeof(int));
	hipMemset(dev_b, 0, BlockSize * sizeof(int));

	// Run sharedReduce first, because b[] is modified in globalReduce

	hipEventRecord(start, 0); //keep start time
	sharedReduceBlockSum << <BlockSize, ThreadSize >> > (dev_b, dev_a);	//calculate
	hipEventRecord(stop, 0); //keep stop time
	hipEventSynchronize(stop); //wait stop event		
	hipEventElapsedTime(&elapsedTime, start, stop);

	hipMemcpy(host_b, dev_b, BlockSize * sizeof(int), hipMemcpyDeviceToHost);

	//Print result
	int answer = (ArraySize + 1)*ArraySize / 2;

	printf("shared:\n");
	sum = 0;
	for (int i = 0; i < BlockSize; i++) {
		sum += host_b[i];
		printf("%d ", host_b[i]);
	}
	printf("sum=%d answer=%d t=%f\n\n", sum, answer, elapsedTime);

	// run globalReduce
	hipEventRecord(start, 0); //keep start time
	globalReduceBlockSum << <BlockSize, ThreadSize >> > (dev_b, dev_a);	//calculate
	hipEventRecord(stop, 0); //keep stop time
	hipEventSynchronize(stop); //wait stop event		
	hipEventElapsedTime(&elapsedTime, start, stop);

	hipMemcpy(host_b, dev_b, BlockSize * sizeof(int), hipMemcpyDeviceToHost);

	//Print result
	printf("global:\n");
	sum = 0;
	for (int i = 0; i < BlockSize; i++) {
		sum += host_b[i];
		printf("%d ", host_b[i]);
	}
	printf("sum=%d answer=%d t=%f\n\n", sum, answer, elapsedTime);
	//hipDeviceSynchronize();
	getchar();

	hipFree(dev_a);
	hipFree(dev_b);
	return 0;
}
