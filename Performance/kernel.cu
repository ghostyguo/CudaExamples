#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "cooperative_groups_helpers.h"
#include <iostream>
#include <memory>
#include <string>
#include <Windows.h>

const int repeat = 10000; //���ƭp�⦸��;
const int arraySize = 1024 * 1024; //������host�Pgpu�O����j�p
BOOL WINAPI QueryPerformanceCounter(_Out_ LARGE_INTEGER *lpPerformanceCount);
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
void addWithCpu(int *c, const int *a, const int *b, unsigned int size);
__global__ void addKernel(int *c, const int *a, const int *b, int size);


// data array for test, global or local static in heap to prevent stck overflow
/*
int a[arraySize];
int b[arraySize];
int c[arraySize];
int d[arraySize];
*/
int main()
{
	// data array for test,  global or local static in heap to prevent stck overflow
	static int a[arraySize];
	static int b[arraySize];
	static int c[arraySize];
	static int d[arraySize];

	// setup performance measure from windows ----------
	LARGE_INTEGER frequency;        // ticks per second
	LARGE_INTEGER t1, t2;           // ticks
	float elapsedTime;

	// setup performance measure from windows ---------
	QueryPerformanceFrequency(&frequency);

	// setup performance meter from CUDA ----------
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	// init array ----------
	for (int i = 0; i < arraySize; i++) {
		a[i] = i;
		b[i] = arraySize + i;
	}

	/// Add by CPU ----------
	QueryPerformanceCounter(&t1); //keep start time
	for (int i = 0; i < repeat; i++) {
		addWithCpu(c, a, b, arraySize);
	}
	QueryPerformanceCounter(&t2); //keep stop time
	elapsedTime = (t2.QuadPart - t1.QuadPart) * 1000.0 / frequency.QuadPart;
	printf("c[%d]=%d, cpu t=%f\n",
		arraySize - 1, c[arraySize - 1], elapsedTime);
	

	// Add by CUDA ----------
	hipEventRecord(start, 0); //keep start time
	hipError_t cudaStatus = addWithCuda(d, a, b, arraySize);
	hipEventRecord(stop, 0); //keep stop time
	hipEventSynchronize(stop); //wait stop event
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}
	hipEventElapsedTime(&elapsedTime, start, stop);	
	printf("d[%d]=%d, gpu t=%f\n",
		arraySize - 1, c[arraySize - 1], elapsedTime);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	getchar();
	return 0;
}

// Add with CPU ---------
void addWithCpu(int *c, const int *a, const int *b, unsigned int size)
{
	for (unsigned int i = 0; i < size; i++) {
		c[i] = a[i] + b[i];
	}
}

// Add with GPU ---------
__global__ void addKernel(int *c, const int *a, const int *b, int size)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < size) {
		c[i] = a[i] + b[i];
	}
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	int dev = 0;
	cudaStatus = hipSetDevice(dev);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	hipSetDevice(dev);

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	/*
	int block = 1;
	unsigned int thread_x = (size-1) / blockDim.x+1;
	unsigned int thread_y = (size-1) % blockDim.x+1;
	dim3 thread = {thread_x, thread_y, 1 };
	*/
	int block = (size - 1) / 1024 + 1;
	int thread = (size>1024) ? 1024 : (size - 1);

	for (int i = 0; i < repeat; i++) {
		addKernel << <block, thread >> > (dev_c, dev_a, dev_b, size);
	}

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}