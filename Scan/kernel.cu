#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#define BlockSize	1
#define ThreadSize	19
#define ArraySize	(BlockSize*ThreadSize)

__device__ void __syncthreads();
__global__ void scanHillisSteele(int *b, int *a)
{
	__shared__ int x[ThreadSize];

	int id = blockIdx.x * blockDim.x + threadIdx.x;
	x[threadIdx.x] = a[id];
	__syncthreads(); //wait copy compelete

	for (int d = 1; d<blockDim.x; d <<= 1)
	{
		if (threadIdx.x >= d) {
			x[threadIdx.x] += x[threadIdx.x - d];
		} //keep 
		__syncthreads();
	}

	b[threadIdx.x] = x[threadIdx.x]; 
}

int main()
{
	int host_a[ArraySize];
	int host_b[ArraySize];
	int *dev_a = 0;
	int *dev_b = 0;
	int sum = 0;
	float elapsedTime;

	// setup performance meter from CUDA ----------
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipSetDevice(0);

	hipMalloc((void**)&dev_a, ArraySize * sizeof(int));
	for (int i = 0; i < ArraySize; i++)
		host_a[i] = i + 1;
	hipMemcpy(dev_a, host_a, ArraySize * sizeof(int), hipMemcpyHostToDevice);

	hipMalloc((void**)&dev_b, ArraySize * sizeof(int));
	//hipMemset(dev_b, 0, ArraySize * sizeof(int));

	// Run scanHillisSteele

	hipEventRecord(start, 0); //keep start time
	scanHillisSteele << <BlockSize, ThreadSize >> > (dev_b, dev_a);	//calculate
	hipEventRecord(stop, 0); //keep stop time
	hipEventSynchronize(stop); //wait stop event		
	hipEventElapsedTime(&elapsedTime, start, stop);

	hipMemcpy(host_b, dev_b, ArraySize * sizeof(int), hipMemcpyDeviceToHost);

	//Print result
	printf("pdf:\n");
	for (int i = 0; i < ArraySize; i++) {
		printf("%4d ", host_a[i]);
	}
	printf("\n");

	printf("cdf:\n");
	for (int i = 0; i < ArraySize; i++) {
		printf("%4d ", host_b[i]);
	}
	printf("\nt=%f\n\n", elapsedTime);




	//hipDeviceSynchronize();
	getchar();

	hipFree(dev_a);
	hipFree(dev_b);
	return 0;
}
